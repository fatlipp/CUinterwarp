#include "hip/hip_runtime.h"
#include "main.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <string>
#include <fstream>
#include <iostream>

#ifndef checkCudaErrors
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        const char *errorStr = NULL;
        fprintf(stderr,
                "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
                "line %i.\n",
                err, errorStr, file, line);
        exit(EXIT_FAILURE);
    }
}
#endif

__global__ void Do(int *data, int size)
{
    data[threadIdx.x] = threadIdx.x;
}

void Alloc()
{
    std::cout << "Alloc start Cuda" << std::endl;
    int *mem;
    hipMalloc((void **)&mem, sizeof(int) * 1024);

    Do<<<1, 1024>>>(mem, 1024);
    checkCudaErrors(hipDeviceSynchronize());

    int *mem2 = (int *)malloc(sizeof(int) * 1024);
    checkCudaErrors(hipMemcpy(mem2, mem, sizeof(int) * 1024, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "" << mem2[0] << " - " << mem2[1] << " - " << mem2[2] << std::endl;

    std::cout << "Alloc end Cuda" << std::endl;
}
